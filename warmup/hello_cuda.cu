#include "hip/hip_runtime.h"
#include <>
#include <device_atomic_functions.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void example_kernel(int example_input)
{
    __shared__ int test_atomics = 1;
    atomicAdd(&test_atomics, 1);
    printf("[%u, %u] hello from cuda; input was %d\n", blockIdx.x, threadIdx.x, example_input);
}

int main(int argc, char** argv)
{
    printf("warmup run\n");
    example_kernel<<<4, 4>>>(argc == 2 ? atoi(argv[1]) : 1);
    int output = 0;
    if(hipError_t e = hipStreamSynchronize(NULL))
    {
        printf("cuda error: %s\n", hipGetErrorString(e));
        output = (int)e;
    }
    printf("no errors\n");
    fflush(stdout);
    return output;
}
