
#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void example_kernel(int example_input)
{
    printf("[%u, %u] hello from cuda; input was %d\n", blockIdx.x, threadIdx.x, example_input);
}

int main(int argc, char** argv)
{
    printf("warmup run\n");
    example_kernel<<<4, 4>>>(1);
    int output = 0;
    if(hipError_t e = hipStreamSynchronize(NULL))
    {
        printf("cuda error: %s\n", hipGetErrorString(e));
        output = (int)e;
    }
    printf("no errors\n");
    fflush(stdout);
    return output;
}
