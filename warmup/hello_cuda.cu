#include "hip/hip_runtime.h"
#include <>
#include <device_atomic_functions.h>
#include <stdio.h>
#include <stdlib.h>
#include "src/util.h"

__global__ void example_kernel(int example_input)
{
    __shared__ int test_atomics;
    if(threadIdx.x == 0) {
        test_atomics = 1;
    }
    __syncthreads();
    atomicAdd(&test_atomics, 1);
    printf("[%u, %u] hello from cuda; input was %d\n", blockIdx.x, threadIdx.x, example_input);
}

MAIN(argc, argv)
{
    printf("warmup run\n");
    example_kernel<<<4, 4>>>(argc == 2 ? atoi(argv[1]) : 1);
    int output = 0;
    if(hipError_t e = hipStreamSynchronize(NULL))
    {
        printf("cuda error: %s\n", hipGetErrorString(e));
        output = (int)e;
    }
    printf("no errors\n");

    fflush(stdout);
    return output;
}
